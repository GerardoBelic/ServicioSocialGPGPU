#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <cmath>
#include <chrono>

#include <cstdio>

#define GLM_ENABLE_EXPERIMENTAL
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"

#include "CLI/CLI.hpp"

struct Computation_Info
{

	Computation_Info(float _timeStep, unsigned _numIterations, unsigned _workGroupSize) :
					 timeStep(_timeStep), numIterations(_numIterations), workGroupSize(_workGroupSize) { }

	float timeStep;
	unsigned numIterations;
	
	unsigned workGroupSize;
	
};

struct Particle_Set
{

    /**
	A particle must store its currect position and velocty.
    */
    Particle_Set(unsigned _numParticles, float _mass = 1.0f) : numParticles(_numParticles), mass(_mass)
    {
		/// Allocate in pinned memory
		unsigned size_memory = numParticles * sizeof(glm::vec3);

		hipHostMalloc((void **)&positions, size_memory);
		hipHostMalloc((void **)&velocities, size_memory);
		
        std::random_device rd;
        std::mt19937 e2(rd());
        std::uniform_real_distribution<float> dist(-1000.0f, 1000.0f);

        for (unsigned i = 0; i < numParticles; ++i)
        {
            (positions + i)->x = dist(e2);
            (positions + i)->y = dist(e2);
            (positions + i)->z = dist(e2);
			
			(velocities + i)->x = 0.0f;
            (velocities + i)->y = 0.0f;
            (velocities + i)->z = 0.0f;
        }
    }
	
	~Particle_Set()
	{
		//hipHostFree(positions);
		//hipHostFree(velocities);
	}

    glm::vec3* positions = nullptr;
    glm::vec3* velocities = nullptr;

    const int numParticles;
    const float mass;

};

__global__ void n_body_vel_calc(glm::vec3 *positions, glm::vec3 *velocities,
								  unsigned numParticles, float mass, float deltaTime)
{
	
	/// Shared memory between a thread group
	/// The size is equal to the number of threads in a group;
	/// this implies that size==blockDim
	__shared__ glm::vec3 temp_tile[256];
	

    unsigned i = blockDim.x * blockIdx.x + threadIdx.x;

    //if (i >= numParticles)
        //return;

    float G = 6.6743e-11f;
	
	glm::vec3 cur_position = positions[i];
	glm::vec3 force(0.0f, 0.0f, 0.0f);

    for (unsigned tile = 0; tile < numParticles; tile += 256)
    {
		temp_tile[threadIdx.x] = positions[tile + threadIdx.x];
		__syncthreads();

		for (unsigned j = 0; j < 256; ++j)
		{
			//if (i == j || ((tile == (numParticles - 1) / blockDim.x) && numParticles % j < ))
			//if (i == j || (tile + j) >= numParticles)
				//continue;
			if (i == j)
				continue;

			float distance2 = glm::distance2(cur_position, temp_tile[j]);
			glm::vec3 direction = glm::normalize(temp_tile[j] - cur_position);

			force += G * mass * mass * direction / distance2;
		}
		
		//__syncthreads();
        
    }
	
	glm::vec3 acceleration = force / mass;

    velocities[i] += acceleration * deltaTime;
	
}

__global__ void n_body_pos_calc(glm::vec3 *positions, glm::vec3 *velocities,
								  unsigned numParticles, float deltaTime)
{
	unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i >= numParticles)
        return;
		
	positions[i] += velocities[i] * deltaTime;
}

auto parse_arguments(int argc, char **argv) -> std::tuple<Particle_Set, Computation_Info>
{
	/// Parse arguments to form the particle set and the computation info
	CLI::App app{"N-Body serial simulation"};

	unsigned numParticles = 0;
	float mass = 1.0f;
	float dt = 1.0f;
	unsigned iterations = 0;
	unsigned workgroupSize = 32;

    app.add_option("--particles", numParticles, "Particle count") -> required();
	app.add_option("--mass", mass, "Mass of each particle (def=1.0[kg])");
	app.add_option("--dt", dt, "Timestep between each iteration (def=1.0[s])");
	app.add_option("--iterations", iterations, "Number of iterations of the simulation") -> required();
	app.add_option("--workgroup_size", workgroupSize, "Number of threads per workgroup (def=32)");
	

    //CLI11_PARSE(app, argc, argv);
    try {
        app.parse(argc, argv);
    } catch (const CLI::ParseError &e) {
        std::exit(app.exit(e));
    }

    std::cout << "Compute info:" << "\n";
    std::cout << "\tParticle count: " << numParticles << " particles" << "\n";
	std::cout << "\tParticle mass: " << mass << " [kg]" << "\n";
	std::cout << "\tTimestep: " << dt << " [s]" << "\n";
	std::cout << "\tIterations: " << iterations << " steps" << "\n";
	std::cout << "\tWorkgroup size: " << workgroupSize << " threads" << "\n\n";

	/// Form the particle set and the computation info
	Particle_Set my_set{numParticles, mass};
	Computation_Info info{dt, iterations, workgroupSize};

	return {my_set, info};

}

int main(int argc, char **argv)
{

	auto [particle_set, computation_info] = parse_arguments(argc, argv);
	
	/// Variables parsed from program arguments
	unsigned numParticles = particle_set.numParticles;
	float mass = particle_set.mass;
	float timeStep = computation_info.timeStep;
	unsigned numIterations = computation_info.numIterations;
	//unsigned workgroupSize = computation_info.workGroupSize;
	unsigned workgroupSize = 256;
	
	/// Kernel variables for dispatching
	dim3 threadsPerBlock = workgroupSize;
	dim3 blocksPerGrid = ((numParticles + workgroupSize - 1) / workgroupSize);
	
	glm::vec3* d_positions = nullptr;
	glm::vec3* d_velocities = nullptr;
	
	unsigned size_memory = numParticles * sizeof(glm::vec3);
	
	/// Start measuring time
	auto start = std::chrono::steady_clock::now();
	
	hipMalloc((void **)&d_positions, size_memory);
	hipMalloc((void **)&d_velocities, size_memory);
	
	/// Wait for hipMalloc to finish
	hipDeviceSynchronize();
	
	/// Wait for malloc to finish
	auto device_malloc = std::chrono::steady_clock::now();
	
	hipMemcpy(d_positions, particle_set.positions, size_memory, hipMemcpyHostToDevice);
	hipMemcpy(d_velocities, particle_set.velocities, size_memory, hipMemcpyHostToDevice);
	
	/// Wait for memcpy to finish
	hipDeviceSynchronize();
	
	auto memcpy_to_device = std::chrono::steady_clock::now();
	
	for (unsigned iter = 0; iter < numIterations; ++iter)
	{
		//n_body_vel_calc<<<blocksPerGrid, threadsPerBlock, workgroupSize * sizeof(glm::vec3)>>>(d_positions, d_velocities, numParticles, mass, timeStep);
		//n_body_pos_calc<<<blocksPerGrid, threadsPerBlock, workgroupSize * sizeof(glm::vec3)>>>(d_positions, d_velocities, numParticles, timeStep);
		n_body_vel_calc<<<blocksPerGrid, threadsPerBlock>>>(d_positions, d_velocities, numParticles, mass, timeStep);
		n_body_pos_calc<<<blocksPerGrid, threadsPerBlock>>>(d_positions, d_velocities, numParticles, timeStep);
	}
	
	/// Wait for computation to finish
	hipDeviceSynchronize();
	
	auto kernel_compute = std::chrono::steady_clock::now();
	
	hipMemcpy(&particle_set.positions[0], d_positions, size_memory, hipMemcpyDeviceToHost);
	hipMemcpy(&particle_set.velocities[0], d_velocities, size_memory, hipMemcpyDeviceToHost);
	
	/// Wait for memcpy to finish
	hipDeviceSynchronize();
	
	/// Finish measuring time
    auto finish = std::chrono::steady_clock::now();
	
	double duration_cuda_malloc = std::chrono::duration_cast<std::chrono::microseconds>(device_malloc - start).count();
	double duration_host_device = std::chrono::duration_cast<std::chrono::microseconds>(memcpy_to_device - device_malloc).count();
	double duration_compute = std::chrono::duration_cast<std::chrono::microseconds>(kernel_compute - memcpy_to_device).count();
	double duration_device_host = std::chrono::duration_cast<std::chrono::microseconds>(finish - kernel_compute).count();

	std::cout << "hipMalloc elapsed time: " << "\n";
    std::cout << "\t" << duration_cuda_malloc << " [us]" << "\n";

    std::cout << "Time to copy memory from host to device: " << "\n";
    std::cout << "\t" << duration_host_device << " [us]" << "\n";

    std::cout << "Compute elapsed time: " << "\n";
    std::cout << "\t" << duration_compute << " [us] (" << duration_compute/1e3 << " [ms]) (" << duration_compute/1e6 << " [s])" << "\n";
	
	std::cout << "Time to copy memory from device back to host: " << "\n";
    std::cout << "\t" << duration_device_host << " [us]" << "\n";
	
	double total_time = duration_cuda_malloc + duration_host_device + duration_compute + duration_device_host;
	
	std::cout << "Total time: " << "\n";
    std::cout << "\t" << total_time << " [us] (" << total_time/1e3 << " [ms]) (" << total_time/1e6 << " [s])" << "\n";

}