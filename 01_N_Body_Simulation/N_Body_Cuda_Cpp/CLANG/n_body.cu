#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <cmath>
#include <chrono>
#include <tuple>

#include "CLI/CLI.hpp"

#include "cuda/hip/hip_vector_types.h"

struct Computation_Info
{

	Computation_Info(float _timeStep, unsigned _numIterations, unsigned _workGroupSize) :
					 timeStep(_timeStep), numIterations(_numIterations), workGroupSize(_workGroupSize) { }

	float timeStep;
	unsigned numIterations;

	unsigned workGroupSize;

};

struct Particle_Set
{

    /**
	A particle must store its currect position and velocty.
    */
    Particle_Set(unsigned _numParticles, float _mass = 1.0f) : positions(_numParticles), velocities(_numParticles), numParticles(_numParticles), mass(_mass)
    {
        std::random_device rd;
        std::mt19937 e2(rd());
        std::uniform_real_distribution<float> dist(-1000.0f, 1000.0f);

        for (auto& pos : positions)
        {
            pos.x = dist(e2);
            pos.y = dist(e2);
            pos.z = dist(e2);
        }

        for (auto& vel : velocities)
        {
            vel = make_float3(0.0f, 0.0f, 0.0f);
        }
    }

    std::vector<float3> positions;
    std::vector<float3> velocities;

    const int numParticles;
    const float mass;

};

__device__ float dist2(float3 A, float3 B)
{
    float3 C = A - B;
    return dot(C, C);
}

__global__ void n_body_vel_calc(float3 *positions, float3 *velocities,
								 unsigned numParticles, float mass, float deltaTime)
{

    unsigned i = blockDim.x * blockIdx.x + threadIdx.x;

    //if (i >= numParticles)
        //return;

    float G = 6.6743e-11f;

	float3 cur_position = positions[i];
	float3 force = make_float3(0.0f, 0.0f, 0.0f);

    for (unsigned j = 0; j < numParticles; ++j)
    {
        if (i == j)
            continue;

		float distance2 = dist2(cur_position, positions[j]);
        float3 direction = normalize(positions[j] - cur_position);

        force += G * mass * mass * direction / distance2;
    }

	float3 acceleration = force / mass;

    velocities[i] += acceleration * deltaTime;

}

__global__ void n_body_pos_calc(float3 *positions, float3 *velocities,
								  unsigned numParticles, float deltaTime)
{
	unsigned i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= numParticles)
        return;

	positions[i] += velocities[i] * deltaTime;
}

auto parse_arguments(int argc, char **argv) -> std::tuple<Particle_Set, Computation_Info>
{
	/// Parse arguments to form the particle set and the computation info
	CLI::App app{"N-Body simulation"};

	unsigned numParticles = 0;
	float mass = 1.0f;
	float dt = 1.0f;
	unsigned iterations = 0;
	unsigned workgroupSize = 32;

    app.add_option("--particles", numParticles, "Particle count") -> required();
	app.add_option("--mass", mass, "Mass of each particle (def=1.0[kg])");
	app.add_option("--dt", dt, "Timestep between each iteration (def=1.0[s])");
	app.add_option("--iterations", iterations, "Number of iterations of the simulation") -> required();
	app.add_option("--workgroup_size", workgroupSize, "Number of threads per workgroup (def=32)");

    //CLI11_PARSE(app, argc, argv);
    try {
        app.parse(argc, argv);
    } catch (const CLI::ParseError &e) {
        std::exit(app.exit(e));
    }

    std::cout << "Compute info:" << "\n";
    std::cout << "\tParticle count: " << numParticles << " particles" << "\n";
	std::cout << "\tParticle mass: " << mass << " [kg]" << "\n";
	std::cout << "\tTimestep: " << dt << " [s]" << "\n";
	std::cout << "\tIterations: " << iterations << " steps" << "\n";
	std::cout << "\tWorkgroup size: " << workgroupSize << " threads" << "\n\n";

	/// Form the particle set and the computation info
	Particle_Set my_set{numParticles, mass};
	Computation_Info info{dt, iterations, workgroupSize};

	return {my_set, info};

}

int main(int argc, char **argv)
{

	auto parse = parse_arguments(argc, argv);
	Particle_Set particle_set = std::get<0>(parse);
	Computation_Info computation_info = std::get<1>(parse);

	/// Variables parsed from program arguments
	unsigned numParticles = particle_set.numParticles;
	float mass = particle_set.mass;
	float timeStep = computation_info.timeStep;
	unsigned numIterations = computation_info.numIterations;
	unsigned workgroupSize = computation_info.workGroupSize;

	/// Kernel variables for dispatching
	dim3 threadsPerBlock(workgroupSize, 1, 1);
	dim3 blocksPerGrid((numParticles + workgroupSize - 1) / workgroupSize, 1, 1);

	float3* d_positions = nullptr;
	float3* d_velocities = nullptr;

	unsigned size_memory = numParticles * sizeof(float3);

	/// Start measuring time
	auto start = std::chrono::steady_clock::now();

	hipMalloc((void **)&d_positions, size_memory);
	hipMalloc((void **)&d_velocities, size_memory);

	/// Wait for hipMalloc to finish
	hipDeviceSynchronize();

	/// Wait for malloc to finish
	auto device_malloc = std::chrono::steady_clock::now();

	hipMemcpy(d_positions, &particle_set.positions[0], size_memory, hipMemcpyHostToDevice);
	hipMemcpy(d_velocities, &particle_set.velocities[0], size_memory, hipMemcpyHostToDevice);

	/// Wait for memcpy to finish
	hipDeviceSynchronize();

	auto memcpy_to_device = std::chrono::steady_clock::now();

	hipEvent_t start_c, stop_c;
	hipEventCreate(&start_c);
	hipEventCreate(&stop_c);
hipEventRecord(start_c);
	for (unsigned iter = 0; iter < numIterations; ++iter)
	{
		n_body_vel_calc<<<blocksPerGrid, threadsPerBlock>>>(d_positions, d_velocities, numParticles, mass, timeStep);
		n_body_pos_calc<<<blocksPerGrid, threadsPerBlock>>>(d_positions, d_velocities, numParticles, timeStep);
	}hipEventRecord(stop_c);
	hipEventSynchronize(stop_c);
float milliseconds = 0;
hipEventElapsedTime(&milliseconds, start_c, stop_c);

std::cout << "Time: " << milliseconds << "\n";
	

	/// Wait for computation to finish
	hipDeviceSynchronize();

	auto kernel_compute = std::chrono::steady_clock::now();

	hipMemcpy(&particle_set.positions[0], d_positions, size_memory, hipMemcpyDeviceToHost);
	hipMemcpy(&particle_set.velocities[0], d_velocities, size_memory, hipMemcpyDeviceToHost);

	/// Wait for memcpy to finish
	hipDeviceSynchronize();

	/// Finish measuring time
    auto finish = std::chrono::steady_clock::now();

	double duration_cuda_malloc = std::chrono::duration_cast<std::chrono::microseconds>(device_malloc - start).count();
	double duration_host_device = std::chrono::duration_cast<std::chrono::microseconds>(memcpy_to_device - device_malloc).count();
	double duration_compute = std::chrono::duration_cast<std::chrono::microseconds>(kernel_compute - memcpy_to_device).count();
	double duration_device_host = std::chrono::duration_cast<std::chrono::microseconds>(finish - kernel_compute).count();

	std::cout << "hipMalloc elapsed time: " << "\n";
    std::cout << "\t" << duration_cuda_malloc << " [us]" << "\n";

    std::cout << "Time to copy memory from host to device: " << "\n";
    std::cout << "\t" << duration_host_device << " [us]" << "\n";

    std::cout << "Compute elapsed time: " << "\n";
    std::cout << "\t" << duration_compute << " [us] (" << duration_compute/1e3 << " [ms]) (" << duration_compute/1e6 << " [s])" << "\n";

	std::cout << "Time to copy memory from device back to host: " << "\n";
    std::cout << "\t" << duration_device_host << " [us]" << "\n";

	double total_time = duration_cuda_malloc + duration_host_device + duration_compute + duration_device_host;

	std::cout << "Total time: " << "\n";
    std::cout << "\t" << total_time << " [us] (" << total_time/1e3 << " [ms]) (" << total_time/1e6 << " [s])" << "\n";

}
