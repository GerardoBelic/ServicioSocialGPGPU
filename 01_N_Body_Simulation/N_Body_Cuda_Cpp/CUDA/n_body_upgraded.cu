#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <cmath>
#include <chrono>

#include "CLI/CLI.hpp"

#include "hip/hip_vector_types.h"

struct Computation_Info
{

	Computation_Info(float _timeStep, unsigned _numIterations, unsigned _workGroupSize) :
					 timeStep(_timeStep), numIterations(_numIterations), workGroupSize(_workGroupSize) { }

	float timeStep;
	unsigned numIterations;

	unsigned workGroupSize;

};

struct Particle_Set
{

    /**
	A particle must store its currect position and velocty.
    */
    Particle_Set(unsigned _numParticles, float _mass = 1.0f) : numParticles(_numParticles), mass(_mass)
    {

        unsigned size_memory = numParticles * sizeof(float3);

        hipHostMalloc((void**)&positions, size_memory);
        hipHostMalloc((void**)&velocities, size_memory);

        std::random_device rd;
        std::mt19937 e2(rd());
        std::uniform_real_distribution<float> dist(-1000.0f, 1000.0f);

        for (unsigned i = 0; i < numParticles; ++i)
        {
            positions[i].x = dist(e2);
            positions[i].y = dist(e2);
            positions[i].z = dist(e2);

            velocities[i] = make_float3(0.0f, 0.0f, 0.0f);
        }

    }

    float3* positions;
    float3* velocities;

    const int numParticles;
    const float mass;

};

__device__ float dist2(float3 A, float3 B)
{
    float3 C = A - B;
    return dot(C, C);
}

__global__ void n_body_vel_calc(float3* positions, float3 * velocities, unsigned workgroupSize,
								 unsigned numParticles, float mass, float deltaTime)
{
    /// Shared memory between a thread group
	/// The size is equal to the number of threads in a group;
	/// this implies that size==blockDim
	extern __shared__ float3 temp_tile[];

    unsigned i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= numParticles)
        return;

    const float G = 6.6743e-11f;

	float3 cur_position = positions[i];

	float3 force = make_float3(0.0f, 0.0f, 0.0f);

    for (unsigned tile = 0; tile < numParticles; tile += workgroupSize)
    {
		temp_tile[threadIdx.x] = positions[tile + threadIdx.x];
		__syncthreads();

		for (unsigned j = 0; j < workgroupSize; ++j)
		{
			if (i == j || ((tile + j) >= numParticles))
				continue;

            float3 neighbor_position = temp_tile[j];

			float inv_distance2 = 1.0f / dist2(cur_position, neighbor_position);
			float3 direction = normalize(neighbor_position - cur_position);

			force += G * mass * mass * inv_distance2 * direction;
		}

		__syncthreads();

    }

	float3 acceleration = force / mass;

    velocities[i] += acceleration * deltaTime;

}

__global__ void n_body_pos_calc(float3 *positions, const float3 *velocities,
								  unsigned numParticles, float deltaTime)
{
	unsigned i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= numParticles)
        return;

	positions[i] += velocities[i] * deltaTime;
}

auto parse_arguments(int argc, char **argv) -> std::tuple<Particle_Set, Computation_Info>
{
	/// Parse arguments to form the particle set and the computation info
	CLI::App app{"N-Body serial simulation"};

	unsigned numParticles = 0;
	float mass = 1.0f;
	float dt = 1.0f;
	unsigned iterations = 0;
	unsigned workgroupSize = 32;

    app.add_option("--particles", numParticles, "Particle count") -> required();
	app.add_option("--mass", mass, "Mass of each particle (def=1.0[kg])");
	app.add_option("--dt", dt, "Timestep between each iteration (def=1.0[s])");
	app.add_option("--iterations", iterations, "Number of iterations of the simulation") -> required();
	app.add_option("--workgroup_size", workgroupSize, "Number of threads per workgroup (def=32)");

    //CLI11_PARSE(app, argc, argv);
    try {
        app.parse(argc, argv);
    } catch (const CLI::ParseError &e) {
        std::exit(app.exit(e));
    }

    std::cout << "Compute info:" << "\n";
    std::cout << "\tParticle count: " << numParticles << " particles" << "\n";
	std::cout << "\tParticle mass: " << mass << " [kg]" << "\n";
	std::cout << "\tTimestep: " << dt << " [s]" << "\n";
	std::cout << "\tIterations: " << iterations << " steps" << "\n";
	std::cout << "\tWorkgroup size: " << workgroupSize << " threads" << "\n\n";

	/// Form the particle set and the computation info
	Particle_Set my_set{numParticles, mass};
	Computation_Info info{dt, iterations, workgroupSize};

	return {my_set, info};

}

int main(int argc, char **argv)
{

	auto [particle_set, computation_info] = parse_arguments(argc, argv);

	/// Variables parsed from program arguments
	unsigned numParticles = particle_set.numParticles;
	float mass = particle_set.mass;
	float timeStep = computation_info.timeStep;
	unsigned numIterations = computation_info.numIterations;
	unsigned workgroupSize = computation_info.workGroupSize;

	/// Kernel variables for dispatching
	dim3 threadsPerBlock = workgroupSize;
	dim3 blocksPerGrid = ((numParticles + workgroupSize - 1) / workgroupSize);

	float3* d_positions = nullptr;
	float3* d_velocities = nullptr;

	unsigned size_memory = numParticles * sizeof(float3);

	/// This is only to initialize the CUDA context and measure the start time correctly
	hipDeviceSynchronize();

	/// Start measuring time
	auto start = std::chrono::steady_clock::now();

	hipMalloc((void **)&d_positions, size_memory);
	hipMalloc((void **)&d_velocities, size_memory);

	/// Wait for hipMalloc to finish
	hipDeviceSynchronize();

	/// Wait for malloc to finish
	auto device_malloc = std::chrono::steady_clock::now();

	hipMemcpy(d_positions, particle_set.positions, size_memory, hipMemcpyHostToDevice);
	hipMemcpy(d_velocities, particle_set.velocities, size_memory, hipMemcpyHostToDevice);

	/// Wait for memcpy to finish
	hipDeviceSynchronize();

	auto memcpy_to_device = std::chrono::steady_clock::now();

	for (unsigned iter = 0; iter < numIterations; ++iter)
	{
		n_body_vel_calc<<<blocksPerGrid, threadsPerBlock, workgroupSize * sizeof(float3)>>>(d_positions, d_velocities, workgroupSize, numParticles, mass, timeStep);
		n_body_pos_calc<<<blocksPerGrid, threadsPerBlock>>>(d_positions, d_velocities, numParticles, timeStep);
	}

	/// Wait for computation to finish
	hipDeviceSynchronize();

	auto kernel_compute = std::chrono::steady_clock::now();

	hipMemcpy(&particle_set.positions[0], d_positions, size_memory, hipMemcpyDeviceToHost);
	hipMemcpy(&particle_set.velocities[0], d_velocities, size_memory, hipMemcpyDeviceToHost);

	/// Wait for memcpy to finish
	hipDeviceSynchronize();

	/// Finish measuring time
    auto finish = std::chrono::steady_clock::now();

	double duration_cuda_malloc = std::chrono::duration_cast<std::chrono::microseconds>(device_malloc - start).count();
	double duration_host_device = std::chrono::duration_cast<std::chrono::microseconds>(memcpy_to_device - device_malloc).count();
	double duration_compute = std::chrono::duration_cast<std::chrono::microseconds>(kernel_compute - memcpy_to_device).count();
	double duration_device_host = std::chrono::duration_cast<std::chrono::microseconds>(finish - kernel_compute).count();

	std::cout << "hipMalloc elapsed time: " << "\n";
    std::cout << "\t" << duration_cuda_malloc << " [us]" << "\n";

    std::cout << "Time to copy memory from host to device: " << "\n";
    std::cout << "\t" << duration_host_device << " [us]" << "\n";

    std::cout << "Compute elapsed time: " << "\n";
    std::cout << "\t" << duration_compute << " [us] (" << duration_compute/1e3 << " [ms]) (" << duration_compute/1e6 << " [s])" << "\n";

	std::cout << "Time to copy memory from device back to host: " << "\n";
    std::cout << "\t" << duration_device_host << " [us]" << "\n";

	double total_time = duration_cuda_malloc + duration_host_device + duration_compute + duration_device_host;

	std::cout << "Total time: " << "\n";
    std::cout << "\t" << total_time << " [us] (" << total_time/1e3 << " [ms]) (" << total_time/1e6 << " [s])" << "\n";

}
