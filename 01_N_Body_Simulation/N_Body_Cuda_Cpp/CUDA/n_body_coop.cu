#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <cmath>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>

#include "CLI/CLI.hpp"

#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper function CUDA error checking and initialization
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

struct Computation_Info
{

	Computation_Info(float _timeStep, unsigned _numIterations, unsigned _workGroupSize) :
					 timeStep(_timeStep), numIterations(_numIterations), workGroupSize(_workGroupSize) { }

	float timeStep;
	unsigned numIterations;

	unsigned workGroupSize;

};

struct Particle_Set
{

    /**
	A particle must store its currect position and velocty.
    */
    Particle_Set(unsigned _numParticles, float _mass = 1.0f) : positions(_numParticles), velocities(_numParticles), numParticles(_numParticles), mass(_mass)
    {
        std::random_device rd;
        std::mt19937 e2(rd());
        std::uniform_real_distribution<float> dist(-1000.0f, 1000.0f);

        for (auto& pos : positions)
        {
            pos.x = dist(e2);
            pos.y = dist(e2);
            pos.z = dist(e2);
        }
    }

    std::vector<float3> positions;
    std::vector<float3> velocities;

    const int numParticles;
    const float mass;

};

__device__ float dist2(float3 A, float3 B)
{
    float3 C = A - B;
    return dot(C, C);
}

__global__ void n_body_calc(float3 *positions, float3 *velocities,
								 unsigned numParticles, float mass, float deltaTime)
{
    cg::grid_group g = cg::this_grid();

    unsigned i = g.thread_rank();

    if (i >= numParticles)
        return;

    float G = 6.6743e-11f;

	float3 cur_position = positions[i];
	float3 force = make_float3(0.0, 0.0, 0.0);

    for (unsigned j = 0; j < numParticles; ++j)
    {
        if (i == j)
            continue;

		float distance2 = dist2(cur_position, positions[j]);
        float3 direction = normalize(positions[j] - cur_position);

        force += G * mass * mass * direction / distance2;
    }

	float3 acceleration = force / mass;

    velocities[i] += acceleration * deltaTime;

    /// Sync
    cg::sync(g);

    positions[i] += velocities[i] * deltaTime;

    /// Sync again
    cg::sync(g);


}

auto parse_arguments(int argc, char **argv) -> std::tuple<Particle_Set, Computation_Info>
{
	/// Parse arguments to form the particle set and the computation info
	CLI::App app{"N-Body serial simulation"};

	unsigned numParticles = 0;
	float mass = 1.0f;
	float dt = 1.0f;
	unsigned iterations = 0;
	unsigned workgroupSize = 32;

    app.add_option("--particles", numParticles, "Particle count") -> required();
	app.add_option("--mass", mass, "Mass of each particle (def=1.0[kg])");
	app.add_option("--dt", dt, "Timestep between each iteration (def=1.0[s])");
	app.add_option("--iterations", iterations, "Number of iterations of the simulation") -> required();
	app.add_option("--workgroup_size", workgroupSize, "Number of threads per workgroup (def=32)");

    //CLI11_PARSE(app, argc, argv);
    try {
        app.parse(argc, argv);
    } catch (const CLI::ParseError &e) {
        std::exit(app.exit(e));
    }

    std::cout << "Compute info:" << "\n";
    std::cout << "\tParticle count: " << numParticles << " particles" << "\n";
	std::cout << "\tParticle mass: " << mass << " [kg]" << "\n";
	std::cout << "\tTimestep: " << dt << " [s]" << "\n";
	std::cout << "\tIterations: " << iterations << " steps" << "\n";
	std::cout << "\tWorkgroup size: " << workgroupSize << " threads" << "\n\n";

	/// Form the particle set and the computation info
	Particle_Set my_set{numParticles, mass};
	Computation_Info info{dt, iterations, workgroupSize};

	return {my_set, info};

}

int main(int argc, char **argv)
{

	auto [particle_set, computation_info] = parse_arguments(argc, argv);

	/// Variables parsed from program arguments
	unsigned numParticles = particle_set.numParticles;
	float mass = particle_set.mass;
	float timeStep = computation_info.timeStep;
	unsigned numIterations = computation_info.numIterations;
	unsigned workgroupSize = computation_info.workGroupSize;

	/// Kernel variables for dispatching
	dim3 threadsPerBlock = workgroupSize;
	dim3 blocksPerGrid = ((numParticles + workgroupSize - 1) / workgroupSize);

	float3* d_positions = nullptr;
	float3* d_velocities = nullptr;

	unsigned size_memory = numParticles * sizeof(float3);

	/// Start measuring time
	auto start = std::chrono::steady_clock::now();

	hipMalloc((void **)&d_positions, size_memory);
	hipMalloc((void **)&d_velocities, size_memory);

	/// Wait for hipMalloc to finish
	hipDeviceSynchronize();

	/// Wait for malloc to finish
	auto device_malloc = std::chrono::steady_clock::now();

	hipMemcpy(d_positions, &particle_set.positions[0], size_memory, hipMemcpyHostToDevice);
	hipMemcpy(d_velocities, &particle_set.velocities[0], size_memory, hipMemcpyHostToDevice);

	/// Wait for memcpy to finish
	hipDeviceSynchronize();

	auto memcpy_to_device = std::chrono::steady_clock::now();

	///-----------------------------------------------------------------------------------------
	hipDeviceProp_t deviceProp;
    int devID = findCudaDevice(argc, (const char **)argv);
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

    // This sample requires being run on a device that supports Cooperative Kernel Launch
    if (!deviceProp.cooperativeLaunch)
    {
        printf("\nSelected GPU (%d) does not support Cooperative Kernel Launch, Waiving the run\n", devID);
        exit(EXIT_WAIVED);
    }

    int numSms = deviceProp.multiProcessorCount;

    int numBlocksPerSm = 0;
    checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, n_body_calc, workgroupSize, 0));

    dim3 dimGrid((numParticles + workgroupSize - 1) / workgroupSize, 1, 1);
    //dim3 dimGrid(numSms*numBlocksPerSm, 1, 1);
    dim3 dimBlock(workgroupSize, 1, 1);

    void* kernelArgs[] =
    {
        (void*)&d_positions,
        (void*)&d_velocities,
        (void*)&numParticles,
        (void*)&mass,
        (void*)&timeStep
    };

    ///-----------------------------------------------------------------------------------------


	for (unsigned iter = 0; iter < numIterations; ++iter)
	{
		//n_body_calc<<<blocksPerGrid, threadsPerBlock>>>(d_positions, d_velocities, numParticles, mass, timeStep);
        //checkCudaErrors(hipLaunchCooperativeKernel((void *)n_body_calc, dimGrid, dimBlock, kernelArgs, 0, 0));
        checkCudaErrors(hipLaunchKernel((void *)n_body_calc, dimGrid, dimBlock, kernelArgs, 0, 0));
		/*{
            hipError_t cudaerr = hipDeviceSynchronize();
            if (cudaerr != hipSuccess)
                printf("kernel launch failed with error \"%s\".\n",
                       hipGetErrorString(cudaerr));
        }*/

	}

	/// Wait for computation to finish
	hipDeviceSynchronize();

	auto kernel_compute = std::chrono::steady_clock::now();

	hipMemcpy(&particle_set.positions[0], d_positions, size_memory, hipMemcpyDeviceToHost);
	hipMemcpy(&particle_set.velocities[0], d_velocities, size_memory, hipMemcpyDeviceToHost);

	/// Wait for memcpy to finish
	hipDeviceSynchronize();

	/// Finish measuring time
    auto finish = std::chrono::steady_clock::now();

	double duration_cuda_malloc = std::chrono::duration_cast<std::chrono::microseconds>(device_malloc - start).count();
	double duration_host_device = std::chrono::duration_cast<std::chrono::microseconds>(memcpy_to_device - device_malloc).count();
	double duration_compute = std::chrono::duration_cast<std::chrono::microseconds>(kernel_compute - memcpy_to_device).count();
	double duration_device_host = std::chrono::duration_cast<std::chrono::microseconds>(finish - kernel_compute).count();

	std::cout << "hipMalloc elapsed time: " << "\n";
    std::cout << "\t" << duration_cuda_malloc << " [us]" << "\n";

    std::cout << "Time to copy memory from host to device: " << "\n";
    std::cout << "\t" << duration_host_device << " [us]" << "\n";

    std::cout << "Compute elapsed time: " << "\n";
    std::cout << "\t" << duration_compute << " [us] (" << duration_compute/1e3 << " [ms]) (" << duration_compute/1e6 << " [s])" << "\n";

	std::cout << "Time to copy memory from device back to host: " << "\n";
    std::cout << "\t" << duration_device_host << " [us]" << "\n";

	double total_time = duration_cuda_malloc + duration_host_device + duration_compute + duration_device_host;

	std::cout << "Total time: " << "\n";
    std::cout << "\t" << total_time << " [us] (" << total_time/1e3 << " [ms]) (" << total_time/1e6 << " [s])" << "\n";

}
