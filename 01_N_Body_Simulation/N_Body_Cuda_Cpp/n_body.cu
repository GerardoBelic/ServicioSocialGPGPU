#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <cmath>
#include <chrono>

#define GLM_ENABLE_EXPERIMENTAL
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"

#include "CLI/CLI.hpp"

struct Computation_Info
{

	Computation_Info(float _timeStep, unsigned _numIterations, unsigned _workGroupSize) :
					 timeStep(_timeStep), numIterations(_numIterations), workGroupSize(_workGroupSize) { }

	float timeStep;
	unsigned numIterations;
	
	unsigned workGroupSize;
	
};

struct Particle_Set
{

    /**
	A particle must store its currect position and velocty.
    */
    Particle_Set(unsigned _numParticles, float _mass = 1.0f) : positions(_numParticles), velocities(_numParticles), numParticles(_numParticles), mass(_mass)
    {
        std::random_device rd;
        std::mt19937 e2(rd());
        std::uniform_real_distribution<float> dist(-1000.0f, 1000.0f);

        for (auto& pos : positions)
        {
            pos.x = dist(e2);
            pos.y = dist(e2);
            pos.z = dist(e2);
        }
    }

    std::vector<glm::vec3> positions;
    std::vector<glm::vec3> velocities;

    const int numParticles;
    const float mass;

};

__global__ void iteracionNCuerpos(glm::vec3 *posiciones, glm::vec3 *velocidades,
								  int numParticulas, float masa, float deltaTiempo)
{

	glm::vec3 fuerza(0.0f, 0.0f, 0.0f);

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= numParticulas)
        return;

    float G = 6.6743e-11f;

    for (int j = 0; j < numParticulas; ++j)
    {

        if (i == j)
            continue;

		float distancia2 = glm::distance2(posiciones[i], posiciones[j]);
        glm::vec3 direccion = glm::normalize(posiciones[j] - posiciones[i]);

        fuerza += G * masa * masa * direccion / distancia2;
        

    }
	
	glm::vec3 aceleracion = fuerza / masa;

    velocidades[i] += aceleracion * deltaTiempo;
	
}

__global__ void integracionNCuerpos(glm::vec3 *posiciones, glm::vec3 *velocidades,
								  int numParticulas, float deltaTiempo)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i >= numParticulas)
        return;
		
	posiciones[i] += velocidades[i] * deltaTiempo;
}

auto parse_arguments(int argc, char **argv) -> std::tuple<Particle_Set, Computation_Info>
{
	/// Parse arguments to form the particle set and the computation info
	CLI::App app{"N-Body serial simulation"};

	unsigned numParticles = 0;
	float mass = 1.0f;
	float dt = 1.0f;
	unsigned iterations = 0;
	unsigned workgroupSize = 32;

    app.add_option("--particles", numParticles, "Particle count") -> required();
	app.add_option("--mass", mass, "Mass of each particle (def=1.0[kg])");
	app.add_option("--dt", dt, "Timestep between each iteration (def=1.0[s])");
	app.add_option("--iterations", iterations, "Number of iterations of the simulation") -> required();
	app.add_option("--workgroup_size", workgroupSize, "Number of threads per workgroup (def=32)");

    //CLI11_PARSE(app, argc, argv);
    try {
        app.parse(argc, argv);
    } catch (const CLI::ParseError &e) {
        std::exit(app.exit(e));
    }

    std::cout << "Compute info:" << "\n";
    std::cout << "\tParticle count: " << numParticles << " particles" << "\n";
	std::cout << "\tParticle mass: " << mass << " [kg]" << "\n";
	std::cout << "\tTimestep: " << dt << " [s]" << "\n";
	std::cout << "\tIterations: " << iterations << " steps" << "\n";
	std::cout << "\tWorkgroup size: " << workgroupSize << " threads" << "\n\n";

	/// Form the particle set and the computation info
	Particle_Set my_set{numParticles, mass};
	Computation_Info info{dt, iterations, workgroupSize};

	return {my_set, info};

}

int main(int argc, char **argv)
{

	auto [particle_set, computation_info] = parse_arguments(argc, argv);
	
	/// Variables parsed from program arguments
	unsigned numParticles = particle_set.numParticles;
	float mass = particle_set.mass;
	float timeStep = computation_info.timeStep;
	unsigned numIterations = computation_info.numIterations;
	unsigned workgroupSize = computation_info.workGroupSize;
	
	/// Kernel variables for dispatching
	dim3 threadsPerBlock = workgroupSize;
	dim3 blocksPerGrid = ((numParticles + workgroupSize - 1) / workgroupSize);
	
	glm::vec3* d_positions = nullptr;
	glm::vec3* d_velocities = nullptr;
	
	unsigned size_memory = numParticles * sizeof(glm::vec3);
	
	/// Start measuring time
	auto start = std::chrono::steady_clock::now();
	
	hipMalloc((void **)&d_positions, size_memory);
	hipMalloc((void **)&d_velocities, size_memory);
	
	hipMemcpy(d_positions, &particle_set.positions[0], size_memory, hipMemcpyHostToDevice);
	hipMemcpy(d_velocities, &particle_set.velocities[0], size_memory, hipMemcpyHostToDevice);
	
	auto memcpy_to_device = std::chrono::steady_clock::now();
	
	for (unsigned iter = 0; iter < numIterations; ++iter)
	{
		iteracionNCuerpos<<<blocksPerGrid, threadsPerBlock>>>(d_positions, d_velocities, numParticles, mass, timeStep);
		integracionNCuerpos<<<blocksPerGrid, threadsPerBlock>>>(d_positions, d_velocities, numParticles, timeStep);
	}
	
	auto kernel_compute = std::chrono::steady_clock::now();
	
	hipMemcpy(&particle_set.positions[0], d_positions, size_memory, hipMemcpyDeviceToHost);
	hipMemcpy(&particle_set.velocities[0], d_velocities, size_memory, hipMemcpyDeviceToHost);
	
	/// Finish measuring time
    auto finish = std::chrono::steady_clock::now();
	
	double duration_host_device = std::chrono::duration_cast<std::chrono::microseconds>(memcpy_to_device - start).count();
	double duration_compute = std::chrono::duration_cast<std::chrono::microseconds>(kernel_compute - memcpy_to_device).count();
	double duration_device_host = std::chrono::duration_cast<std::chrono::microseconds>(finish - kernel_compute).count();

    std::cout << "Time to copy memory from host to device: " << "\n";
    std::cout << "\t" << duration_host_device << " [us]" << "\n";

    std::cout << "Compute elapsed time: " << "\n";
    std::cout << "\t" << duration_compute << " [us]" << "\n";
	
	std::cout << "Time to copy memory from device back to host: " << "\n";
    std::cout << "\t" << duration_device_host << " [us]" << "\n";
	
	std::cout << "Total time: " << "\n";
    std::cout << "\t" << duration_host_device + duration_compute + duration_device_host << " [us]" << "\n";

}