#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <cmath>

#define GLM_ENABLE_EXPERIMENTAL
//#define GLM_FORCE_CUDA
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"

// __device__ float dist2()
// {


    
// }

struct Particulas
{

    /**

    Una particula debe guardar su posicion y velocidad actual
    Se agrega el vector de aceleracion/fuerza para no tener que crear

    */
    Particulas(int _numParticulas, float _masa = 1.0f) : posiciones(_numParticulas), velocidades(_numParticulas), numParticulas(_numParticulas), masa(_masa)
    {
        std::random_device rd;
        std::mt19937 e2(rd());
        std::uniform_real_distribution<float> dist(-1000.0f, 1000.0f);

        for (auto& posicion : posiciones)
        {
            posicion.x = dist(e2);
            posicion.y = dist(e2);
            posicion.z = dist(e2);
        }
    }

    std::vector<glm::vec3> posiciones;
    std::vector<glm::vec3> velocidades;

    const int numParticulas;
    const float masa;

};

__global__ void iteracionNCuerpos(glm::vec3 *posiciones, glm::vec3 *velocidades,
								  int numParticulas, float masa, float deltaTiempo)
{

	glm::vec3 fuerza(0.0f, 0.0f, 0.0f);

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= numParticulas)
        return;

    float G = 6.6743e-11f;

    for (int j = 0; j < numParticulas; ++j)
    {

        if (i == j)
            continue;

		float distancia2 = glm::distance2(posiciones[i], posiciones[j]);
        glm::vec3 direccion = glm::normalize(posiciones[j] - posiciones[i]);

        fuerza += G * masa * masa * direccion / distancia2;
        

    }
	
	glm::vec3 aceleracion = fuerza / masa;

    velocidades[i] += aceleracion * deltaTiempo;
	
	
	
	
    

}

__global__ void integracionNCuerpos(glm::vec3 *posiciones, glm::vec3 *velocidades,
								  int numParticulas, float deltaTiempo)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i >= numParticulas)
        return;
		
	posiciones[i] += velocidades[i] * deltaTiempo;
}

int main()
{

	int numParticulas = 1024 * 2;
	float masa = 1.0e9f;
	float deltaTiempo = 1.0f;

	Particulas set_1(numParticulas, masa);
	
	glm::vec3* d_posiciones = nullptr;
	glm::vec3* d_velocidades = nullptr;
	
	int size_memory = numParticulas * sizeof(glm::vec3);
	
	hipMalloc((void **)&d_posiciones, size_memory);
	hipMalloc((void **)&d_velocidades, size_memory);
	
	hipMemcpy(d_posiciones, &set_1.posiciones[0], size_memory, hipMemcpyHostToDevice);
	hipMemcpy(d_velocidades, &set_1.velocidades[0], size_memory, hipMemcpyHostToDevice);

	dim3 threadsPerBlock(32);
	dim3 blocksPerGrid((set_1.numParticulas + 31) / 32);
	
	std::cout << set_1.posiciones[0].x << " " << set_1.posiciones[0].y << " " << set_1.posiciones[0].z << std::endl;
	
	for (int i = 0; i < 2000; ++i)
	{
		iteracionNCuerpos<<<blocksPerGrid, threadsPerBlock>>>(d_posiciones, d_velocidades, numParticulas, masa, deltaTiempo);
		integracionNCuerpos<<<blocksPerGrid, threadsPerBlock>>>(d_posiciones, d_velocidades, numParticulas, deltaTiempo);
		hipMemcpy(&set_1.posiciones[0], d_posiciones, size_memory, hipMemcpyDeviceToHost);
		std::cout << set_1.posiciones[0].x << " " << set_1.posiciones[0].y << " " << set_1.posiciones[0].z << std::endl;
	}
	
	//hipMemcpy(&set_1.posiciones[0], d_posiciones, size_memory, hipMemcpyDeviceToHost);
	
	


}